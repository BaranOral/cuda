#include "hip/hip_runtime.h"
%%writefile hw1.cu


#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) y[i] = a*x[i] + y[i];
    
}

__host__ 
void getInformationAboutSystem(){
    int deviceNumber;
    hipGetDeviceCount(&deviceNumber);
   
    for (int i = 0; i< deviceNumber; i++){
   
        hipDeviceProp_t deviceProp;
        hipError_t err =  hipGetDeviceProperties (&deviceProp, i);
        if (!err) {
            printf("The device name: %s \n", deviceProp.name );
            printf("the maximum number of thread blocks: %d \n", deviceProp.maxThreadsPerBlock);    
            printf("the maximum number of threads per block at the beginning: %d \n", deviceProp.maxBlocksPerMultiProcessor);
            }
    }
}
__host__ 
float* generateRandomElements(int N, float constant){

    float *arr;
    arr = (float*)malloc(N*sizeof(float));  
    if ( arr == NULL ){
        printf("Run out of memmory!\n");
        exit(1);
    }

    
    for (int i = 0; i<N; i++){
          arr[i] = ((float)rand()/RAND_MAX)* constant;
          printf("index %d: %f | ", i , arr[i]);
          
    }
    return arr;
}

int main(void)
{
    
    int N;
    float A;
    printf("Enter a size for array: ");
    scanf("%d", &N);


    printf("Enter a scalar value: ");
    scanf("%f", &A);
    // printf("%f, %d", A, N);
    
    // getInformationAboutSystem();
    

    float *x, *y, *d_x, *d_y;

    printf("X Vector is createad as: \n");
    x = generateRandomElements(N, 1.0f);
    printf("\n--------------------------------------\n");
    printf("Y Vector is createad as: \n");
    y = generateRandomElements(N, 2.0f);
    printf("\n--------------------------------------\n");

    
    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    saxpy<<<N,1>>>(N, A, d_x, d_y);

    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    // float maxError = 0.0f;
    printf("After SAXPY Y Vector is like following\n");
    for (int i = 0; i < N; i++)
        printf("index %d: %f | ", i, y[i]);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}   
